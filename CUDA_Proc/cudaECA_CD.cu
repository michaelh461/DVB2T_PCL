#include "hip/hip_runtime.h"
#include "RCF.h"
#include "cudaECA.cuh"
#include <stdio.h>
#include <iostream>
#include <string>
#include <fstream>
#include <math.h>
#include "ARD_Makers.h"

#define real_prod(a, b, c, d) (a*c - b*d)
#define comp_prod(a, b, c, d) (a*d + b*c)

using namespace std;

__global__ void ECA_CD(double * RefSymbolData, double * SurvSymbolData, double * D, double * SurvSymbols_Cancelled){
   
    const int nSymbols = 60;
    const int nCarriers = 27841;

    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = (int)nCarriers/(blockDim.x * gridDim.x);
 
    int start = index*stride;
    int stop = start + stride;
    if(stop > nCarriers){
        stop = nCarriers;
    }

    for(int k = start; k < stop; k++){
        // Reference channel carrier amplitudes (Q)
        double Q[2][nSymbols];
        // Surveillance channel carrier amplitudes (Y)
        double Y[2][nSymbols];
        for(int i = 0; i < nSymbols; i++){
            Q[0][i] = RefSymbolData[2*k*nSymbols + 2*i];
            Q[1][i] = RefSymbolData[2*k*nSymbols + 2*i + 1];
            Y[0][i] = SurvSymbolData[2*k*nSymbols + 2*i];
            Y[1][i] = SurvSymbolData[2*k*nSymbols + 2*i + 1];
        }
        
        // Form X matrix (clutter subspace matrix)
        double X[3][2][nSymbols];
        
        //X[0] = D' * Q
        for(int i = 0; i < nSymbols; i++){
            X[0][0][i] = real_prod(D[2*i], -D[2*i + 1], Q[0][i], Q[1][i]);
            X[0][1][i] = comp_prod(D[2*i], -D[2*i + 1], Q[0][i], Q[1][i]);
        }
        
        //X[1] = Q
        for(int i = 0; i < nSymbols; i++){
            X[1][0][i] =  Q[0][i];
            X[1][1][i] =  Q[1][i];
        }
    
        //X[2] = D * Q
        for(int i = 0; i < nSymbols; i++){
            X[2][0][i] = real_prod(D[2*i], D[2*i + 1], Q[0][i], Q[1][i]);
            X[2][1][i] = comp_prod(D[2*i], D[2*i + 1], Q[0][i], Q[1][i]);
        }    
        
        //Form A matrix; A = X'*X   
        double A[2][3][3];
        for(int i = 0; i < 3; i++){
            for(int j = 0; j < 3; j++){
                A[0][i][j] = 0;
                A[1][i][j] = 0; 
                for(int n = 0; n < nSymbols; n++){
                    A[0][i][j] = A[0][i][j] + real_prod(X[j][0][n], -X[j][1][n], X[i][0][n], X[i][1][n]);
		    if(i == j){
			A[1][i][j] = 0;
		    }
		    else{
                        A[1][i][j] = A[1][i][j] + comp_prod(X[j][0][n], -X[j][1][n], X[i][0][n], X[i][1][n]);
		   }
                }                 
            }
        }
        
        // Invert A matrix, C = A^(-1)
        double C[2][3][3];
        InvertMatrix_3x3(A, C);
        
        //Form B matrix, B = X'*Y
        double B[2][3];
        for(int i = 0; i < 3; i++){
            B[0][i] = 0; 
            B[1][i] = 0;
            for(int n = 0; n < nSymbols; n++){
                B[0][i] = B[0][i] + X[i][0][n]*Y[0][n] + X[i][1][n]*Y[1][n];
                B[1][i] = B[1][i] + X[i][0][n]*Y[1][n] - X[i][1][n]*Y[0][n];
            }
        }
        
        //Calculate F = A*B = (C.')*B
        double F[2][3];
        for(int i = 0; i < 3; i++){
            F[0][i] = 0; 
            F[1][i] = 0;
            for(int n = 0; n < 3; n++){
                F[0][i] = F[0][i] + C[0][n][i]*B[0][n] - C[1][n][i]*B[1][n];
                F[1][i] = F[1][i] + C[0][n][i]*B[1][n] + C[1][n][i]*B[0][n];
            }
        }
        
        // Allocate memory for real and complex parts of cancelled data
        double Z[2][nSymbols];

        // Perform ECA on carrier
        for(int i = 0; i < nSymbols; i++){
            Z[0][i] = Y[0][i];
            Z[1][i] = Y[1][i];
            for(int n = 0; n < 3; n++){
                Z[0][i] = Z[0][i] - (X[n][0][i]*F[0][n] - X[n][1][i]*F[1][n]);
                Z[1][i] = Z[1][i] - (X[n][0][i]*F[1][n] + X[n][1][i]*F[0][n]);
            }
        }
        
       // Copy data to be returned
        for(int i = 0; i < nSymbols; i++){
            SurvSymbols_Cancelled[2*k*nSymbols + 2*i] = Z[0][i];
            SurvSymbols_Cancelled[2*k*nSymbols + 2*i + 1] = Z[1][i];
        }
    }    

}

int main(void){
	// File containing RCF data
	string fileName = "../frames/symbol_data.rcf";

	// Create RCF object and read file header
	cRCF * oRCF = new cRCF();
	oRCF->readHeader(fileName, true);

	// Read data from file
	uint64_t nSamples = oRCF->getNSamples();
	oRCF->readData(fileName, 0, nSamples);

	// Get pointers to reference and surveillance samples
	float * RefData = oRCF->getReferenceArrayFloatPointer();
	float * SurvData = oRCF->getSurveillanceArrayFloatPointer();

	cout << "RCF read complete" << endl;
    
    const int nCarriers = 27841;
    const int nSymbols = 60;
    
    double * RefSymbolData;
    hipMallocManaged(&RefSymbolData, 2*nSymbols*nCarriers*sizeof(double));
    double * SurvSymbolData;
    hipMallocManaged(&SurvSymbolData, 2*nSymbols*nCarriers*sizeof(double));
    
    for(int i = 0; i < nSamples; i++){
        RefSymbolData[i*2] = (double)RefData[2*i];
        RefSymbolData[i*2+1] = (double)RefData[2*i+1];
        SurvSymbolData[i*2] = (double)SurvData[2*i];
        SurvSymbolData[i*2+1] = (double)SurvData[2*i+1];
    }

    //Form doppler-shift matrix D
    double * D;
    hipMallocManaged(&D, 2*nSymbols*sizeof(double));
    // Note: these must be parametized
    double Fd = 0.75;
    double T = 7/64e6;
    double Ts = nCarriers*T;
    const long double pi = 3.14159265358979323846264338328L;
    for(int i = 0; i < nSymbols; i++){
        D[2*i] = cos(2*pi*Fd*i*Ts);
        D[2*i + 1] = sin(2*pi*Fd*i*Ts);
    }

    double * Surv_Cancelled;
    hipMallocManaged(&Surv_Cancelled, 2*nSymbols*nCarriers*sizeof(double));
    
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	std::cout << numSMs << std::endl;
    // Execute ECA_CD cuda core
    int n = 16*numSMs;
    ECA_CD<<<6, 256>>>(RefSymbolData, SurvSymbolData, D, Surv_Cancelled);
    hipDeviceSynchronize();

    for(int i = 0; i < nSamples; i++){
        SurvData[2*i] = Surv_Cancelled[i*2];
        SurvData[2*i+1] = Surv_Cancelled[i*2 + 1];
    }

        std::ofstream ofs;
         ofs.open("../frames/cancelled_rcf.rcf");
        ofs << (*oRCF);


        return 0;



//    hipFree(RefSymbolData);
//    hipFree(SurvSymbolData);
//    hipFree(D);
//    hipFree(Surv_Cancelled);
    
//    ofstream ofs;
//    ofs.open("../frames/new_rcf.rcf");
//    ofs << (*oRCF);

    return 0;
}

// Utilities

/*  CofactorMatrix_3x3:
 *  Returns the matrix of cofactors of the 3x3 COMPLEX matrix A
 *  A[0] = 3x3 real data 
 *  A[1] = 3x3 imag data
 *  i.e. returns C such that adj(A) = C^(T)
 */
__device__ void CofactorMatrix_3x3(double A[][3][3], double C[][3][3]){
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 3; j++){
            // Real component
            C[0][i][j] = (
                      (A[0][(i+1)%3][(j+1)%3] * A[0][(i+2)%3][(j+2)%3] 
                    -  A[1][(i+1)%3][(j+1)%3] * A[1][(i+2)%3][(j+2)%3]) 
                    - (A[0][(i+1)%3][(j+2)%3] * A[0][(i+2)%3][(j+1)%3] 
                    -  A[1][(i+1)%3][(j+2)%3] * A[1][(i+2)%3][(j+1)%3])
                    );
            
            // Complex Component
            C[1][i][j] = (
                      (A[0][(i+1)%3][(j+1)%3]*A[1][(i+2)%3][(j+2)%3] 
                    +  A[1][(i+1)%3][(j+1)%3]*A[0][(i+2)%3][(j+2)%3]) 
                    - (A[0][(i+1)%3][(j+2)%3]*A[1][(i+2)%3][(j+1)%3] 
                    +  A[1][(i+1)%3][(j+2)%3]*A[0][(i+2)%3][(j+1)%3])
                    );
        }
    }
}

/* Determinant_3x3:
 * Returns the determinant of the 3x3 COMPLEX matrix A
 * C = pre-computed matrix of cofactors
 * A[0], C[0] = 3x3 real data
 * A[1], C[1] = 3x3 imag data
 * det[0] = real component of determinant 
 * det[1] = imag component of determinant
 */
__device__ void Determinant_3x3(double A[][3][3], double C[][3][3], double det[2]){
    det[0] = 0;
    det[1] = 1;
         
    for(int i = 0; i < 3; i++){
        det[0] = det[0] + real_prod(A[0][0][i], A[1][0][i], 
                                    C[0][0][i], C[1][0][i]);
        
        det[1] = det[1] + comp_prod(A[0][0][i], A[1][0][i], 
                                    C[0][0][i], C[1][0][i]);
    }
}

/* InvertMatrix_3x3:
 * Returns the inverse of the 3x3 COMPLEX matrix A
 * A[0], B[0] = real data
 * A[1], B[1] = imag data
 * B = A^(-1)
 */
__device__ void InvertMatrix_3x3(double A[][3][3], double B[][3][3]){
    double C [2][3][3];
    double det[2];
    CofactorMatrix_3x3(A, C);
    Determinant_3x3(A, C, det);
    // Calculate B = adj(A)/det(A) = C^(T)/det(A)
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < 3; j++){
            B[0][i][j] = ((C[0][i][j]*det[0] - C[1][i][j]*det[1])/
                          (det[0]*det[0] + det[1]*det[1]));
            B[1][i][j] = -((C[0][i][j]*det[1] + C[1][i][j]*det[0])/
                           (det[0]*det[0] + det[1]*det[1]));
         }
    }
}
